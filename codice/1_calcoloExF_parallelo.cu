#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <thrust/device_vector.h>
#include <thrust/system/system_error.h>
#include <vector>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <vector>
#include <math.h>
#include <algorithm>
#include <functional>
#include <map>
#include <omp.h>
#include <chrono>
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    printf("%s\n",hipGetErrorString(x)); \
    system("pause"); \
    return EXIT_FAILURE;}} while(0)
static void HandleError(hipError_t err, const char*file,int line){ 
if (err != hipSuccess) {
 printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
 exit( EXIT_FAILURE ); }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
using namespace std;

/* funzione che sostituisce ogni valore del vettore di input con la loro posizione con ripetizione (stesso valore = stesso indice)
   per avere nodi con nomi da 1 ad n con n= numero di nodi del grafo */
vector<int> changeVect(vector<int> input, int N) {

	map<int, int> ranks; // creo un contenitore associativo chiave-valore per evitare di creare un vettore con dimensione = max valore nell'input

	int rank = 1;

	for (int index = 0; index < N; index++) { // riempio ranks associando a ogni valore la sua posizione

		int element = input[index];

		if (ranks[element] == 0) // aumento il rank solo se non si tratta di un valore ripetuto
		{
			ranks[element] = rank;
			rank++;
		}
	}

	for (int index = 0; index < N; index++) // sostituisco ogni valore del vettore di input con il relativo rank associato
	{
		int element = input[index];
		input[index] = ranks[input[index]];
	}

	return input;

}

/* funzione che apre il file di input .txt in formato SNAP e ricava i vettori IC e IR necessari per il calcolo dell'ExF e restituisce un intero che è il grado max nella rete*/
int calcoloICIR(string filename, vector <int>& IC, vector <int>& IR) {

	ifstream grafo_snap;  // apro il file contenente il grafo in formato SNAP
	grafo_snap.open(filename, ios::in);

	vector<int> c1;  // creo i vettori che conterranno i dati
	vector<int> c2;

	vector< pair <int, int> > vect; // creo vettore di coppie

	// controllo che il file esista
	if (!grafo_snap) {
		cout << "File non esistente";
	}
	else {

		int nodo_partenza; // dichiaro due int di supporto equivalenti a testa e coda dell'arco
		int nodo_arrivo;

		while (grafo_snap >> nodo_partenza && grafo_snap >> nodo_arrivo) {  // salvo il primo e il secondo elemento di ogni riga nei vettori
			vect.push_back(make_pair(nodo_partenza, nodo_arrivo));

			if (grafo_snap.eof()) // controllo che il ciclo scandisca il file fino al termine
				break;
		}
	}
	grafo_snap.close();

	sort(vect.begin(), vect.end());

	for (auto it = std::make_move_iterator(vect.begin()),
		end = std::make_move_iterator(vect.end()); it != end; ++it)
	{
		c1.push_back(std::move(it->first));
		c2.push_back(std::move(it->second));
	}

	vector<int> vect_union; // dichiaro il vettore unione per poter calcore il "rank"
	vect_union.reserve(c1.size() + c2.size()); // alloco la memoria per il vettore unione
	vect_union.insert(vect_union.end(), c1.begin(), c1.end()); // faccio l'unione dei due vettori
	vect_union.insert(vect_union.end(), c2.begin(), c2.end());

	vector<int> output; // vettore di output da dividire in seguito

	size_t N = vect_union.size();

	output = changeVect(vect_union, N);

	// devo calcolare IC e IR finali per darli in input alla funzione che calcola l'ExF

	// IC è già nella sua forma finale (ovvero = seconda meta' del vettore output), devo solo aggiungere uno 0 in testa
	for (int k = output.size() / 2; k < output.size(); k++) {
		IC.push_back(output[k]);
	}
	IC.insert(IC.begin(), 0);

	// I step per calcolo IR: calcolare il numero di nodi del grafo
	std::vector<int>::iterator it_numNodi;
	it_numNodi = std::max_element(output.begin(), output.end());
	int num_nodi = it_numNodi[0];

	// II step per calcolo IR: considero la prima parte del vettore output, da qui il nome preIR
	vector<int> preIR(output.begin(), output.begin() + output.size() / 2);

	// III step per calcolo IR: calcolo per ogni nodo il numero di ripetizioni (se ci sono)

	int j = 0;
	int i = 0;
	int sum = 1; // parte da 1 perche' ci sara' sempre almeno un nodo 
	int partialSum = 1;
	int maxSum = 1;

	while (j < preIR.size() && i < preIR.size() - 1) {


		if (preIR[i] != preIR[i + 1]) {

			IR.push_back(sum);
			j++;
			maxSum = max(partialSum, maxSum);
			partialSum = 1;
		}
		else {
			partialSum = partialSum + 1;

		}


		sum++;
		i++;



	}


	for (int k = j; k < num_nodi; k++) {
		IR.push_back(sum);
	}

	IR.insert(IR.begin(), 0); // inserisco in testa lo 0 come da definizione
	return  max(partialSum, maxSum);

}


/*
* funzione che calcola il grado dei cluster: ogni cluster è formato da nodo seme e altri due nodi (in questo caso non importa se i nodi sono a distanza 1 o a distanza 2)
* grado cluster=numero di archi che connettono nodi nel cluster con nodi non nel cluster
*/
__device__ int gradoCluster(int* nodi, int* IR, int* IC) {
	// per ogni nodo calcolo il numero di vicini
	int grado = 0;
	for (int a = 0; a < 3; a++) {
		int nodo = nodi[a];//nodo considerato per calcolare il numero di vicini
		int valRiga = IR[nodo] - IR[nodo - 1];//numero di vicini del nodo
		if (valRiga > 0) {//controllo se il numero di vicini è >0 altrimenti il grado è 0
			// printf("Numero di vicini %i:%i\n", nodo, valRiga);
			for (int k = IR[nodo - 1] + 1; k < IR[nodo - 1] + 1 + valRiga; k++) {//scandisco tutti i vicini del nodo considerato e controllo se questi nodi non sono gi? nel cluster
				int vicino = IC[k];
				// printf("Nodo vicino di %i:%i\n", nodo ,vicino);
				if (vicino != nodi[0] && vicino != nodi[1] && vicino != nodi[2]) {// se il vicino del nodo non è nel cluster aumento il grado del cluster
					grado++;
				}

			}
		}
	}

	//printf("Grado del cluster(%i, %i, %i):%i,\n", nodi[0], nodi[1], nodi[2], grado);

	return grado;
}

/*
* kernel che dati IC, IR calcola ExF per ogni nodo del grafo
*/
__global__ void expectedForce(int* IR_vec, int* IC_vec, int n_IR, int gradoMax, double* d_exf)
{

	//indice i per scorrere l'array IR_vec 
	int seed = blockDim.x * blockIdx.x + threadIdx.x;
	if (seed < n_IR && seed != 0) {//controllo di non eccedere la dimensione dell'array e che non venga calcolato ExF del primo elemento di IR_vec
		double ExF = 0; //ogni thread memorizza il valore di ExF in questa variabile
		int indiceGradi = 0;
		int totalFI = 0;//ogni thread memorizza la somma dei gradi in questa variabile
		//printf("Thread %d calcola nodo %d \n", seed, IR_vec[seed]);

		int valRiga = IR_vec[seed];
		int* distOne = new int[gradoMax];//vettore di vicini a distanza 1 dal nodo seed di dimensione gradoMax (numero massimo di vicini di un nodo nel grafo considerato)
		if (distOne == NULL) { printf("distOne failed\n"); return; } // controllo se il puntatore è nullo

		int dist = IR_vec[seed] - IR_vec[seed - 1]; //numero di vicini del nodo seed
		if (dist == 0) { //se il nodo considerato non ha vicini allora ExF=0
			d_exf[seed] = 0;
			//printf("Exf del nodo %i: %f\n", seed, d_exf[seed]);

		}
		else // se il nodo considerato ha almeno un vicino
		{

			//calcolo i nodi a distanza uno dal nodo seme e li metto nel vettore distOne
			int indiceDistOne = 0;
			for (int k = IR_vec[seed - 1] + 1; k <= valRiga; k++) {
				int valB = IC_vec[k];
				//printf("Nodo a distanza uno da %i:%i\n",seed, valB);
				if (valB != seed) {//controllo che il vicino del nodo non sia il nodo stesso 
					distOne[indiceDistOne] = valB;
					indiceDistOne++;
				}
			}
			if (indiceDistOne == 0) {//se a questo punto il vettore distOne è vuoto allora il nodo seme punta solo a se stesso


				d_exf[seed] = 0;
				//printf("Exf del nodo %i: %f\n", seed, d_exf[seed]);

			}
			else {//se il vettore distOne non è vuoto
				//calcolo la dimensione dell'array gradi considerando il caso peggiore: ogni nodo nodo ha gradoMax vicini e tutti i suoi vicini hanno gradoMax vicibi
				int worstCaseSize = gradoMax * gradoMax + gradoMax * (gradoMax - 1);
				int* gradi = new int[worstCaseSize]; //this allocates memory on a local memory runtime heap which has the lifetime of the context
				if (gradi == NULL) { printf("gradi failed\n"); return; }// controllo se il puntatore è nullo

				//devo creare i cluster con 2 nodi a distanza 1 da seed
			// il doppio ciclo for mi permette di trovare le combinazioni di nodi a distanza 1 da seed
				for (int k = 0; k < indiceDistOne - 1; k++) {
					for (int j = k + 1; j < indiceDistOne; j++) {
						int nodiCluster[3];  //vettore che contiene i 3 nodi che formano il cluster considerato
						//printf("Nodi nel cluster:%i,%i,%i\n", seed, distOne[k], distOne[j]);
						nodiCluster[0] = seed;
						nodiCluster[1] = distOne[k];
						nodiCluster[2] = distOne[j];

						int grado = gradoCluster(nodiCluster, IR_vec, IC_vec); //calcolo il grado del cluster considerato
						int mult = 2; //ogni cluster si puo' creare in due combinazioni 

						for (int count = 0; count < mult; count++) { // inserisco nel vettore gradi 2 volte il grado del cluster considerato
							gradi[indiceGradi] = grado;

							indiceGradi++;
						}


						totalFI += mult * grado;

					}
				}


				//devo creare i cluster con un nodo a distanza 1 e 1 a distanza 2 dal nodo seed
				for (int k = 0; k < indiceDistOne; k++) { //scandisco i nodi a distanza 1 dal nodo seed

					int nodo = distOne[k]; //considero un nodo a distanza 1 
					// printf("Secondo nodo cluster:%i\n", nodo);
					 //itero su tutti i nodi a distanza 1 e trovo i loro vicini
					int valRiga = IR_vec[nodo] - IR_vec[nodo - 1]; //numero di vicini del nodo considerato
					if (valRiga > 0) { //se il nodo considerato ha almeno un vicino posso procedere con il calcolo del cluster
						//printf("Numero di vicini di %i:%i\n", nodo, valRiga);
						 //printf("Inizio del for %i ", IR[nodo - 1] + 1);
						for (int j = IR_vec[nodo - 1] + 1; j < IR_vec[nodo - 1] + 1 + valRiga; j++) { //scandisco i vicini del nodo considerato che sono i nodi a distanza 2 del nodo seed
							//nodo vicino del vicino, calcolo il numero di archi uscenti 
							int valB = IC_vec[j];
							int nodiCluster[3];
							// printf("Terzo nodo cluster:%i\n", valB);

							// printf("Nodi nel cluster a dist 1 e 2:%i,%i,%i\n", seed, nodo, valB);
							nodiCluster[0] = seed;
							nodiCluster[1] = nodo;
							nodiCluster[2] = valB;;


							int grado = gradoCluster(nodiCluster, IR_vec, IC_vec); // calcolo il grado del cluster
							gradi[indiceGradi] = grado; //inserisco il grado calcolato nel vettore gradi


							indiceGradi++;

							totalFI += grado;
						}

					}


				}
				delete[]distOne;

				/*
				* Dopo aver calcolato il vettore dei gradi dei cluster controllo :
				* - se la somma totale dei gradi è 0 allora anche ExF=0
				* - altrimenti normalizzo il vettore del gradi (dividendo ogni grado per la somma dei gradi) e calcolo ExF
				*/

				if (totalFI == 0) {

					d_exf[seed] = 0;
					//printf("Exf del nodo %i: %f\n", seed, d_exf[seed]);

				}
				else {

					double norm = 0;

					for (int K = 0; K < indiceGradi; K++) { //normalizzo l'array gradi e calcolo il valore di ExF
						if (gradi[K] != 0) {
							norm = (float)gradi[K] / totalFI;
							ExF -= log(norm) * norm;
						}


					}


					d_exf[seed] = ExF;
					//printf("Exf del nodo %i: %f\n", seed, d_exf[seed]);

				}

				delete[] gradi;

			}

		}


	}



}


int main(int argc, char* argv[])
{
	auto start = chrono::steady_clock::now();
	//definisco le variabili per calcolare l'occupazione della memoria
	size_t free_byte;
	size_t total_byte;

	//la variabile malloc_limit viene inizializzata alla quantità di memoria heap necessaria per il calcolo di ExF 
	//(la memoria heap ci serve per memorizzare il vettore dei vicini a distanza 1 e il vettore dei gradi dei cluster (uno per ogni thread))
	size_t malloc_limit = 1073741824; // 512 MB = 536870912; 1 GB = 1073741824; 256 MB = 268435456;
	HANDLE_ERROR(hipDeviceSetLimit(hipLimitMallocHeapSize, malloc_limit)); //imposto la capienza della memoria heap a mallo_limit
	//printf("dimensione della heap:%zi\n", malloc_limit);

	//creo i vettori h_IC e h_IR nell'host
	vector <int> h_IC;
	vector <int> h_IR;
	string name_file(argv[1]);
	//inizializzo i vettori h_IC e h_IR in base al file .txt di input tramite la funzione calcoloICIR 
	//la funzione calcoloICIR restituisce maxDegree che è un valore che verrà riutilizzato nel kernel per trovare un upper bound alla dimensione degli array distOne
	int maxDegree = calcoloICIR(name_file,h_IC, h_IR);
	//printf("Max degree:%i\n", maxDegree);

	/*cout << "IR = ";
	cout << "[";
	for (int i = 0; i < h_IR.size() - 1; i++) {
		cout << h_IR[i] << ", ";
	}
	cout << h_IR[h_IR.size() - 1] << "]" << endl;


	cout << "IC = ";
	cout << "[";
	for (int i = 0; i < h_IC.size() - 1; i++)
	{
		cout << h_IC[i] << ", ";
	}
	cout << h_IC[h_IC.size() - 1] << "]";*/

	//calcolo la lunghezza dei vettori h_IC e h_IR
	size_t n_IC = h_IC.size();
	size_t n_IR = h_IR.size();


	//per poter passare i vettori h_IC e h_IR al kernel devo creare delle copie (d_IC e d_IR) del tipo device_vector della libreria thrust
	thrust::device_vector<int> d_IR = h_IR;
	thrust::device_vector<int> d_IC = h_IC;

	//creo i puntatori raw che verranno passati al kernel per i vettori h_IC e h_IR 
	int* IR_vec = thrust::raw_pointer_cast(d_IR.data());
	int* IC_vec = thrust::raw_pointer_cast(d_IC.data());


	int N = n_IR;
	//printf("Numero di nodi del grafo:%i\n", N-1);

	size_t size = N * sizeof(double);

	//Creo il vettore h_exf in cui verranno memorizzati i valori ExF di tutti i nodi del grafo e lo alloco nella memoria dell'host
	double* h_exf = (double*)malloc(size);
	memset(h_exf, 0, N);//imposto a 0 gli elementi di h_exf

	//Creo il vettore d_exf che verrà usato dal device per memorizzare i valori di ExF di tutti i nodi del grafo, lo alloco in memoria host e lo copio in memoria device
	double* d_exf;
	HANDLE_ERROR(hipMalloc(&d_exf, size));
	HANDLE_ERROR(hipMemcpy(d_exf, h_exf, size, hipMemcpyHostToDevice));

	//lancio il kernel expectedForce
	int threadsPerBlock = 512;
	int blocksPerGrid = (n_IR + threadsPerBlock - 1) / threadsPerBlock;
	expectedForce << <blocksPerGrid, threadsPerBlock >> > (IR_vec, IC_vec, n_IR, maxDegree, d_exf);
	CUDA_CALL(hipDeviceSynchronize());

	//copio i risultati da d_exf a h_exf e li stampo 
	HANDLE_ERROR(hipMemcpy(h_exf, d_exf, size, hipMemcpyDeviceToHost));
	for (int i = 1; i < n_IR; i++) {
		printf("Exf del nodo %i: %f\n", i, h_exf[i]);
	}

	//libero la memoria allocata
	hipFree(d_exf);
	free(h_exf);

	// show memory usage of GPU
	HANDLE_ERROR(hipMemGetInfo(&free_byte, &total_byte));

	double free_db = (double)free_byte;

	double total_db = (double)total_byte;

	double used_db = total_db - free_db;

	printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",

		used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0, total_db / 1024.0 / 1024.0);



	auto end = chrono::steady_clock::now();
	auto diff = end - start;//calcolo tempo totale di computazione
	cout << chrono::duration <double, milli>(diff).count() << " ms" << endl;

	return 0;
}
